#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define MAX_THREADS_PER_BLOCK 256

__global__ void _vecAdd(float* vec1, float* vec2, float* returnVec,
		int vecSize) {
	int i;

	i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < vecSize) {
		returnVec[i] = vec1[i] + vec2[i];
	}
}

__global__ void _vecMul(float* vec1, float* vec2, float* returnVec,
		int vecSize) {
	int i;

	i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < vecSize) {
		returnVec[i] = vec1[i] * vec2[i];
	}
}

//Device vector add implementation
__host__ float* vecAdd(float* vec1, float* vec2, int vecSize) {
	float *d_vec1, *d_vec2;
	float *d_returnVec;
	float *h_returnVec;
	int vecMemSize;
	int blockWidth;
	int numOfBlocks;

	blockWidth =
			MAX_THREADS_PER_BLOCK < vecSize ? MAX_THREADS_PER_BLOCK : vecSize;
	numOfBlocks = vecSize / blockWidth + (vecSize % blockWidth == 0 ? 0 : 1);

	dim3 dimBlock(blockWidth, 1);
	dim3 dimGrid(numOfBlocks, 1);

	vecMemSize = vecSize * sizeof(float);
	h_returnVec = (float*) malloc(vecMemSize);
	hipMalloc((void**) &d_vec1, vecMemSize);
	hipMalloc((void**) &d_vec2, vecMemSize);
	hipMalloc((void**) &d_returnVec, vecMemSize);

	hipMemcpy(d_vec1, vec1, vecMemSize, hipMemcpyHostToDevice);
	hipMemcpy(d_vec2, vec2, vecMemSize, hipMemcpyHostToDevice);

	_vecAdd<<<dimGrid, dimBlock, 0, 0>>>(d_vec1, d_vec2, d_returnVec, vecSize);
	hipDeviceSynchronize();

	hipError_t err;
	err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Function call failed!\n");
		fprintf(stderr, "(Why?) : %s\n", hipGetErrorString(err));
	}

	hipMemcpy(h_returnVec, d_returnVec, vecMemSize, hipMemcpyDeviceToHost);
	hipFree(d_vec1);
	hipFree(d_vec2);
	hipFree(d_returnVec);

	return h_returnVec;
}

//Device vector mul implementation
__host__ float* vecMul(float* vec1, float* vec2, int vecSize) {
	float *d_vec1, *d_vec2;
	float *d_returnVec;
	float *h_returnVec;
	int vecMemSize;
	int blockWidth;
	int numOfBlocks;

	blockWidth =
			MAX_THREADS_PER_BLOCK < vecSize ? MAX_THREADS_PER_BLOCK : vecSize;
	numOfBlocks = vecSize / blockWidth + (vecSize % blockWidth == 0 ? 0 : 1);

	dim3 dimBlock(blockWidth, 1);
	dim3 dimGrid(numOfBlocks, 1);

	vecMemSize = vecSize * sizeof(float);
	h_returnVec = (float*) malloc(vecMemSize);
	hipMalloc((void**) &d_vec1, vecMemSize);
	hipMalloc((void**) &d_vec2, vecMemSize);
	hipMalloc((void**) &d_returnVec, vecMemSize);

	hipMemcpy(d_vec1, vec1, vecMemSize, hipMemcpyHostToDevice);
	hipMemcpy(d_vec2, vec2, vecMemSize, hipMemcpyHostToDevice);

	_vecMul<<<dimGrid, dimBlock, 0, 0>>>(d_vec1, d_vec2, d_returnVec, vecSize);
	hipDeviceSynchronize();

	hipMemcpy(h_returnVec, d_returnVec, vecMemSize, hipMemcpyDeviceToHost);
	hipFree(d_vec1);
	hipFree(d_vec2);
	hipFree(d_returnVec);

	return h_returnVec;
}

int func_add(float *x, float *y, int sz) {
	int i;
	float *a;
	a = (float *) malloc(sizeof(float) * sz);
	if (!a) {
		printf("memory allocation error\n");
		exit(-1);
	}
	memcpy(a, x, sz * (sizeof(float)));

	x = vecAdd(x, y, sz);

	for (i = 0; i < sz; i++) {
		if (x[i] != a[i] + y[i]) {
			return 0;
		}
	}

	free(a);
	return 1;
}

int func_mul(float *x, float *y, int sz) {
	int i;
	float *a;

	a = (float *) malloc(sizeof(float) * sz);
	if (!a) {
		printf("memory allocation error\n");
		exit(-1);
	}
	memcpy(a, x, sz * (sizeof(float)));

	x = vecMul(x, y, sz);

	for (i = 0; i < sz; i++) {
		if (x[i] != a[i] * y[i]) {
			return 0;
		}
	}

	free(a);
	return 1;
}
